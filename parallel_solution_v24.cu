#include "hip/hip_runtime.h"
//
// Created by phuc on 16/01/2022.
//

#include "parallel_solution_v24.cuh"
#include "timer.cuh"
#include "parallel_solution_v4.cuh"

void ParallelSolutionV24::calculateSeamMap(int32_t *d_inputImage, uint32_t inputWidth, uint32_t inputHeight,
                                           uint32_t blockSize) {
    uint32_t gridSize = ((inputWidth - 1) / blockSize + 1) * (inputHeight - 1);
    uint32_t zero = 0;

    // Create Device Memory
    bool *isBlockFinished;
    CHECK(hipMalloc(&isBlockFinished, gridSize * sizeof(bool)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::blockCount), &zero, sizeof(u_int32_t), 0, hipMemcpyHostToDevice))
    CHECK(hipMemset(isBlockFinished, 0, gridSize * sizeof(bool)))

    // Run Device Methods
    KernelFunction::updateSeamMapKernelPipelining<<<gridSize, blockSize>>>(d_inputImage, inputWidth, isBlockFinished);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    // Copy Memory from Device to Host

    // Free Device Memory
    CHECK(hipFree(isBlockFinished))

    // Free Host Memory

    // Return result

}

PnmImage ParallelSolutionV24::run(const PnmImage &inputImage, int argc, char **argv) {
    // Extract arguments
    int nDeletingSeams = 1;
    dim3 blockSize(32, 32); // Default
    if (argc > 0)
        nDeletingSeams = int(strtol(argv[0], nullptr, 10));
    if (argc > 1) {
        blockSize.x = strtol(argv[1], nullptr, 10);
        blockSize.y = strtol(argv[2], nullptr, 10);
    }

    // Start Timer
    printf("Running Parallel Solution Version 2 + 4 with blockSize=(%d;%d).\n", blockSize.x, blockSize.y);
    GpuTimer timer;
    GpuTimer stepTimer;

    float cal_energy_time = 0;
    float cal_seam_time = 0;
    float extract_seam_time = 0;
    float delete_seam_time = 0;

    timer.Start();

    // Create Host Variable
    PnmImage outputImage(inputImage.getWidth() - nDeletingSeams, inputImage.getHeight());

    // Create Host Memory
    auto *seam = (uint32_t *) malloc(inputImage.getHeight() * sizeof(uint32_t));
    auto *energyMap = (int32_t *) malloc(inputImage.getHeight() * inputImage.getWidth() * sizeof(int32_t));

    // Create Device Memory
    uchar3 *d_inputImage;
    CHECK(hipMalloc(&d_inputImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    uchar3 *d_inputImageTemp;
    CHECK(hipMalloc(&d_inputImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    int32_t *d_grayImage;
    CHECK(hipMalloc(&d_grayImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_grayImageTemp;
    CHECK(hipMalloc(&d_grayImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_energyMap;
    CHECK(hipMalloc(&d_energyMap, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpy(d_inputImage, inputImage.getPixels(),
                     inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3), hipMemcpyHostToDevice))

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::c_filterX), SOBEL_X, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::c_filterY), SOBEL_Y, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))

    // Run Kernel functions
    convertToGrayScale(d_inputImage, inputImage.getWidth(), inputImage.getHeight(), blockSize, d_grayImage);
    for (int i = 0; i < nDeletingSeams; ++i) {
        // 1. Calculate the Energy Map
        stepTimer.Start();
        calculateEnergyMap(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), blockSize, d_energyMap);
        stepTimer.Stop();
        cal_energy_time += stepTimer.Elapsed();

        // 2. Dynamic Programming
        stepTimer.Start();
        calculateSeamMap(d_energyMap, inputImage.getWidth() - i, inputImage.getHeight(), blockSize.x * blockSize.y);
        stepTimer.Stop();
        cal_seam_time += stepTimer.Elapsed();

        // 3. Extract the seam
        stepTimer.Start();
        CHECK(hipMemcpy(energyMap, d_energyMap,
                         (inputImage.getWidth() - i) * inputImage.getHeight() * sizeof(int32_t),
                         hipMemcpyDeviceToHost))
        extractSeam(energyMap, inputImage.getWidth() - i, inputImage.getHeight(), seam);
        stepTimer.Stop();
        extract_seam_time += stepTimer.Elapsed();

        // 4. Delete the seam
        stepTimer.Start();
        deleteSeam(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_grayImageTemp);
        deleteSeam(d_inputImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_inputImageTemp);
        stepTimer.Stop();
        delete_seam_time += stepTimer.Elapsed();

        swap(d_grayImage, d_grayImageTemp);
        swap(d_inputImage, d_inputImageTemp);
    }

    // Copy memory from device to host
    CHECK(hipMemcpy(outputImage.getPixels(), d_inputImage,
                     outputImage.getWidth() * outputImage.getHeight() * sizeof(uchar3), hipMemcpyDeviceToHost))

    // Free Device Memory
    CHECK(hipFree(d_inputImage))
    CHECK(hipFree(d_inputImageTemp))
    CHECK(hipFree(d_grayImage))
    CHECK(hipFree(d_grayImageTemp))
    CHECK(hipFree(d_energyMap))

    // Free Host Memory
    free(seam);
    free(energyMap);

    // Stop Timer
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
    printf("Step time: 1/%.3f ms 2/%.3f ms 3/%.3f ms 4/%.3f ms", cal_energy_time, cal_seam_time, extract_seam_time, delete_seam_time);
    printf("-------------------------------\n");

    // Return
    return outputImage;
}
