#include "hip/hip_runtime.h"
#include "parallel_solution_v234.cuh"
#include "parallel_solution_v2.cuh"
#include "timer.cuh"

void ParallelSolutionV234::calculateEnergyMap(const int32_t *d_inputImage, uint32_t inputWidth, uint32_t inputHeight,
                                              dim3 blockSize, int32_t *d_outputImage) {
    // Create Host Memory
    dim3 gridSize((inputWidth - 1) / blockSize.x + 1, (inputHeight - 1) / blockSize.y + 1);
    size_t smemSize = (blockSize.x + FILTER_SIZE - 1) * (blockSize.y + FILTER_SIZE - 1) * sizeof(int32_t);

    // Create Device Memory
    int32_t *d_outputImageX;
    CHECK(hipMalloc(&d_outputImageX, inputWidth * inputHeight * sizeof(int32_t)))
    int32_t *d_outputImageY;
    CHECK(hipMalloc(&d_outputImageY, inputWidth * inputHeight * sizeof(int32_t)))

    // Copy Memory from Host to Device

    // Run Device Methods
    KernelFunction::convolutionKernel_v2<<<gridSize, blockSize, smemSize>>>(d_inputImage, inputWidth, inputHeight, true, FILTER_SIZE, d_outputImageX);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    KernelFunction::convolutionKernel_v2<<<gridSize, blockSize, smemSize>>>(d_inputImage, inputWidth, inputHeight, false, FILTER_SIZE, d_outputImageY);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    KernelFunction::addAbsKernel<<<gridSize, blockSize>>>(d_outputImageX, d_outputImageY, inputWidth, inputHeight, d_outputImage);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    // Copy Memory from Device to Host

    // Free Device Memory
    CHECK(hipFree(d_outputImageX))
    CHECK(hipFree(d_outputImageY))

    // Free Host Memory

    // Return result
}

PnmImage ParallelSolutionV234::run(const PnmImage &inputImage, int argc, char **argv) {

    // Extract arguments
    int nDeletingSeams = 1;
    dim3 blockSize(32, 32); // Default
    if (argc > 0)
        nDeletingSeams = int(strtol(argv[0], nullptr, 10));
    if (argc > 1) {
        blockSize.x = strtol(argv[1], nullptr, 10);
        blockSize.y = strtol(argv[2], nullptr, 10);
    }

    // Start Timer
    printf("Running Parallel Solution Version 2 + 3 + 4 with blockSize=(%d;%d).\n", blockSize.x, blockSize.y);
    GpuTimer timer;
    GpuTimer stepTimer;

    float cal_energy_time = 0;
    float cal_seam_time = 0;
    float extract_seam_time = 0;
    float delete_seam_time = 0;

    timer.Start();

    // Create Host Variable
    PnmImage outputImage(inputImage.getWidth() - nDeletingSeams, inputImage.getHeight());

    // Create Host Memory
    auto *seam = (uint32_t *) malloc(inputImage.getHeight() * sizeof(uint32_t));
    auto *energyMap = (int32_t *) malloc(inputImage.getHeight() * inputImage.getWidth() * sizeof(int32_t));

    // Create Device Memory
    uchar3 *d_inputImage;
    CHECK(hipMalloc(&d_inputImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    uchar3 *d_inputImageTemp;
    CHECK(hipMalloc(&d_inputImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    int32_t *d_grayImage;
    CHECK(hipMalloc(&d_grayImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_grayImageTemp;
    CHECK(hipMalloc(&d_grayImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_energyMap;
    CHECK(hipMalloc(&d_energyMap, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpy(d_inputImage, inputImage.getPixels(),
                     inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3), hipMemcpyHostToDevice))

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::c_filterX), SOBEL_X, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::c_filterY), SOBEL_Y, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))

    // Run Kernel functions
    convertToGrayScale(d_inputImage, inputImage.getWidth(), inputImage.getHeight(), blockSize, d_grayImage);
    for (int i = 0; i < nDeletingSeams; ++i) {
        // 1. Calculate the Energy Map
        stepTimer.Start();
        calculateEnergyMap(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), blockSize, d_energyMap);
        stepTimer.Stop();
        cal_energy_time += stepTimer.Elapsed();

        // 2. Dynamic Programming
        stepTimer.Start();
        calculateSeamMap(d_energyMap, inputImage.getWidth() - i, inputImage.getHeight(), blockSize.x * blockSize.y);
        stepTimer.Stop();
        cal_seam_time += stepTimer.Elapsed();

        // 3. Extract the seam
        stepTimer.Start();
        CHECK(hipMemcpy(energyMap, d_energyMap,
                         (inputImage.getWidth() - i) * inputImage.getHeight() * sizeof(int32_t),
                         hipMemcpyDeviceToHost))
        extractSeam(energyMap, inputImage.getWidth() - i, inputImage.getHeight(), seam);
        stepTimer.Stop();
        extract_seam_time += stepTimer.Elapsed();

        // 4. Delete the seam
        stepTimer.Start();
        deleteSeam(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_grayImageTemp);
        deleteSeam(d_inputImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_inputImageTemp);
        stepTimer.Stop();
        delete_seam_time += stepTimer.Elapsed();

        swap(d_grayImage, d_grayImageTemp);
        swap(d_inputImage, d_inputImageTemp);
    }

    // Copy memory from device to host
    CHECK(hipMemcpy(outputImage.getPixels(), d_inputImage,
                     outputImage.getWidth() * outputImage.getHeight() * sizeof(uchar3), hipMemcpyDeviceToHost))

    // Free Device Memory
    CHECK(hipFree(d_inputImage))
    CHECK(hipFree(d_inputImageTemp))
    CHECK(hipFree(d_grayImage))
    CHECK(hipFree(d_grayImageTemp))
    CHECK(hipFree(d_energyMap))

    // Free Host Memory
    free(seam);
    free(energyMap);

    // Stop Timer
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
    printf("Step time: 1) %.3f ms \t 2) %.3f ms \t 3) %.3f ms \t 4) %.3f ms\n", cal_energy_time, cal_seam_time, extract_seam_time, delete_seam_time);
    printf("-------------------------------\n");

    // Return
    return outputImage;
}
