#include "hip/hip_runtime.h"
#include "parallel_solution_v234.cuh"
#include "parallel_solution_v2.cuh"
#include "timer.cuh"

void ParallelSolutionV234::calculateEnergyMap(const int32_t *d_inputImage, uint32_t inputWidth, uint32_t inputHeight,
                                              dim3 blockSize, int32_t *d_outputImage) {
    // Create Host Memory
    dim3 gridSize((inputWidth - 1) / blockSize.x + 1, (inputHeight - 1) / blockSize.y + 1);
    size_t smemSize = (blockSize.x + FILTER_SIZE - 1) * (blockSize.y + FILTER_SIZE - 1) * sizeof(int32_t);

    // Create Device Memory
    int32_t *d_outputImageX;
    CHECK(hipMalloc(&d_outputImageX, inputWidth * inputHeight * sizeof(int32_t)))
    int32_t *d_outputImageY;
    CHECK(hipMalloc(&d_outputImageY, inputWidth * inputHeight * sizeof(int32_t)))

    // Copy Memory from Host to Device

    // Run Device Methods
    KernelFunction::convolutionKernel_v2<<<gridSize, blockSize, smemSize>>>(d_inputImage, inputWidth, inputHeight, true, FILTER_SIZE, d_outputImageX);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    KernelFunction::convolutionKernel_v2<<<gridSize, blockSize, smemSize>>>(d_inputImage, inputWidth, inputHeight, false, FILTER_SIZE, d_outputImageY);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    KernelFunction::addAbsKernel<<<gridSize, blockSize>>>(d_outputImageX, d_outputImageY, inputWidth, inputHeight, d_outputImage);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    // Copy Memory from Device to Host

    // Free Device Memory
    CHECK(hipFree(d_outputImageX))
    CHECK(hipFree(d_outputImageY))

    // Free Host Memory

    // Return result
}

PnmImage ParallelSolutionV234::run(const PnmImage &inputImage, int argc, char **argv) {

    // Extract arguments
    int nDeletingSeams = 1;
    dim3 blockSize(32, 32); // Default
    if (argc > 0)
        nDeletingSeams = int(strtol(argv[0], nullptr, 10));
    if (argc > 1) {
        blockSize.x = strtol(argv[1], nullptr, 10);
        blockSize.y = strtol(argv[2], nullptr, 10);
    }

    // Start Timer
    printf("Running Parallel Solution Version 2 + 3 + 4 with blockSize=(%d;%d).\n", blockSize.x, blockSize.y);
    GpuTimer timer;
    timer.Start();

    // Create Host Variable
    PnmImage outputImage(inputImage.getWidth() - nDeletingSeams, inputImage.getHeight());

    // Create Host Memory
    auto *seam = (uint32_t *) malloc(inputImage.getHeight() * sizeof(uint32_t));
    auto *energyMap = (int32_t *) malloc(inputImage.getHeight() * inputImage.getWidth() * sizeof(int32_t));

    // Create Device Memory
    uchar3 *d_inputImage;
    CHECK(hipMalloc(&d_inputImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    uchar3 *d_inputImageTemp;
    CHECK(hipMalloc(&d_inputImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    int32_t *d_grayImage;
    CHECK(hipMalloc(&d_grayImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_grayImageTemp;
    CHECK(hipMalloc(&d_grayImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_energyMap;
    CHECK(hipMalloc(&d_energyMap, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpy(d_inputImage, inputImage.getPixels(),
                     inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3), hipMemcpyHostToDevice))

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::c_filterX), SOBEL_X, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::c_filterY), SOBEL_Y, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))

    // Run Kernel functions
    convertToGrayScale(d_inputImage, inputImage.getWidth(), inputImage.getHeight(), blockSize, d_grayImage);
    for (int i = 0; i < nDeletingSeams; ++i) {
        // 1. Calculate the Energy Map
        calculateEnergyMap(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), blockSize, d_energyMap);
        // 2. Dynamic Programming
        calculateSeamMap(d_energyMap, inputImage.getWidth() - i, inputImage.getHeight(), blockSize.x * blockSize.y);
        // 3. Extract the seam
        CHECK(hipMemcpy(energyMap, d_energyMap,
                         (inputImage.getWidth() - i) * inputImage.getHeight() * sizeof(int32_t),
                         hipMemcpyDeviceToHost))
        extractSeam(energyMap, inputImage.getWidth() - i, inputImage.getHeight(), seam);
        // 4. Delete the seam
        deleteSeam(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_grayImageTemp);
        deleteSeam(d_inputImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_inputImageTemp);
        swap(d_grayImage, d_grayImageTemp);
        swap(d_inputImage, d_inputImageTemp);
    }

    // Copy memory from device to host
    CHECK(hipMemcpy(outputImage.getPixels(), d_inputImage,
                     outputImage.getWidth() * outputImage.getHeight() * sizeof(uchar3), hipMemcpyDeviceToHost))

    // Free Device Memory
    CHECK(hipFree(d_inputImage))
    CHECK(hipFree(d_inputImageTemp))
    CHECK(hipFree(d_grayImage))
    CHECK(hipFree(d_grayImageTemp))
    CHECK(hipFree(d_energyMap))

    // Free Host Memory
    free(seam);
    free(energyMap);

    // Stop Timer
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
    printf("-------------------------------\n");

    // Return
    return outputImage;
}
