#include "hip/hip_runtime.h"
#include "parallel_solution_v3.cuh"
#include "timer.cuh"

namespace KernelFunction {
    __global__ void
    updateSeamMapKernelBackward(int32_t *input, u_int32_t inputWidth,
                                int32_t currentRow) {
        u_int32_t c = blockIdx.x * blockDim.x + threadIdx.x;
        if (c < inputWidth) {
            int32_t minVal = input[convertIndex(currentRow + 1, c, inputWidth)];
            if (c > 0)
                minVal = min(minVal, input[convertIndex(currentRow + 1, c - 1, inputWidth)]);
            if (c + 1 < inputWidth)
                minVal = min(minVal, input[convertIndex(currentRow + 1, c + 1, inputWidth)]);
            input[convertIndex(currentRow, c, inputWidth)] += minVal;
        }
    }
}

IntImage ParallelSolutionV3::calculateSeamMap(const IntImage &inputImage, uint32_t blockSize) {
    // Create Host Memory
    uint32_t gridSize = (inputImage.getWidth() - 1) / blockSize + 1;
    IntImage outputImage = IntImage(inputImage.getWidth(), inputImage.getHeight());

    // Create Device Memory
    int32_t *d_inputImage;
    CHECK(hipMalloc(&d_inputImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpy(d_inputImage, inputImage.getPixels(),
                     inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t), hipMemcpyHostToDevice))

    // Run Device Methods
    hipStream_t streamForward, streamBackward;
    hipStreamCreate(&streamForward);
    hipStreamCreate(&streamBackward);
    for (int i = 1; i <= inputImage.getHeight() / 2; ++i) {
        // Forward
        KernelFunction::updateSeamMapKernel<<<gridSize, blockSize, 0, streamForward>>>(d_inputImage, inputImage.getWidth(), i);
        // Backward
        if (int(inputImage.getHeight()) - i - 1 > inputImage.getHeight() / 2) {
            KernelFunction::updateSeamMapKernelBackward<<<gridSize, blockSize, 0, streamBackward>>>(d_inputImage, inputImage.getWidth(),
                    int(inputImage.getHeight()) - i - 1);
        }
        hipStreamSynchronize(streamForward);
        hipStreamSynchronize(streamBackward);
        CHECK(hipGetLastError())
    }
    hipStreamDestroy(streamForward);
    hipStreamDestroy(streamBackward);

    // Copy Memory from Device to Host
    CHECK(hipMemcpy(outputImage.getPixels(), d_inputImage,
                     outputImage.getWidth() * outputImage.getHeight() * sizeof(int32_t), hipMemcpyDeviceToHost))

    // Free Device Memory
    CHECK(hipFree(d_inputImage))

    // Free Host Memory

    // Return result
    return outputImage;
}

PnmImage ParallelSolutionV3::run(const PnmImage &inputImage, int argc, char **argv) {
    // Extract arguments
    int nDeletingSeams = 1;
    dim3 blockSize(32, 32); // Default
    if (argc > 0)
        nDeletingSeams = int(strtol(argv[0], nullptr, 10));
    if (argc > 1) {
        blockSize.x = strtol(argv[1], nullptr, 10);
        blockSize.y = strtol(argv[2], nullptr, 10);
    }
    printf("Running Parallel Solution Version 3 with blockSize=(%d;%d).\n", blockSize.x, blockSize.y);
    GpuTimer timer;
    timer.Start();

    PnmImage outputImage = inputImage;
    for (int i = 0; i < nDeletingSeams; ++i) {
        // 1. Convert to GrayScale
        IntImage grayImage = convertToGrayScale(outputImage, blockSize);
        // 2. Calculate the Energy Map
        IntImage energyMap = calculateEnergyMap(grayImage, blockSize);
        // 3. Dynamic Programming
        IntImage seamMap = calculateSeamMap(energyMap, blockSize.x * blockSize.y);
        // 4. Extract the seam
        auto *seam = (uint32_t *) malloc(energyMap.getHeight() * sizeof(uint32_t));
        extractSeam(seamMap, seam);
        // 5. Delete the seam
        outputImage = deleteSeam(outputImage, seam);
        free(seam);
    }
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
    printf("-------------------------------\n");
    return outputImage;
}

void ParallelSolutionV3::extractSeam(const IntImage &energyMap, uint32_t *seam) {
    // Find minSeam
    u_int32_t minValCol1 = 0;
    u_int32_t minValCol2 = 0;
    u_int32_t middleRow = energyMap.getHeight() / 2;
    int32_t bestVal = energyMap.getPixels()[KernelFunction::convertIndex(middleRow, 0, energyMap.getWidth())] +
                      energyMap.getPixels()[KernelFunction::convertIndex(middleRow + 1, 0, energyMap.getWidth())];

    for (int c = 0; c < energyMap.getWidth(); ++c) {
        if (energyMap.getPixels()[KernelFunction::convertIndex(middleRow, c, energyMap.getWidth())] +
            energyMap.getPixels()[KernelFunction::convertIndex(middleRow + 1, c, energyMap.getWidth())]
            < bestVal) {
            bestVal = energyMap.getPixels()[KernelFunction::convertIndex(middleRow, c, energyMap.getWidth())] +
                      energyMap.getPixels()[KernelFunction::convertIndex(middleRow + 1, c, energyMap.getWidth())];
            minValCol1 = c;
            minValCol2 = c;
        }

        if (c > 0 &&
            energyMap.getPixels()[KernelFunction::convertIndex(middleRow, c - 1, energyMap.getWidth())] +
            energyMap.getPixels()[KernelFunction::convertIndex(middleRow + 1, c, energyMap.getWidth())]
            <= bestVal) {
            bestVal = energyMap.getPixels()[KernelFunction::convertIndex(middleRow, c - 1, energyMap.getWidth())] +
                      energyMap.getPixels()[KernelFunction::convertIndex(middleRow + 1, c, energyMap.getWidth())];
            minValCol1 = c - 1;
            minValCol2 = c;
        }

        if (c + 1 < energyMap.getWidth() &&
            energyMap.getPixels()[KernelFunction::convertIndex(middleRow, c + 1, energyMap.getWidth())] +
            energyMap.getPixels()[KernelFunction::convertIndex(middleRow + 1, c, energyMap.getWidth())]
            < bestVal) {
            bestVal = energyMap.getPixels()[KernelFunction::convertIndex(middleRow, c + 1, energyMap.getWidth())] +
                      energyMap.getPixels()[KernelFunction::convertIndex(middleRow + 1, c, energyMap.getWidth())];
            minValCol1 = c + 1;
            minValCol2 = c;
        }
    }
    // Trace back
    seam[energyMap.getHeight() / 2] = minValCol1;
    seam[energyMap.getHeight() / 2 + 1] = minValCol2;

    for (int r = int(energyMap.getHeight() / 2 - 1); r >= 0; --r) {
        auto c = minValCol1;
        if (c > 0) {
            if (energyMap.getPixels()[KernelFunction::convertIndex(r, c - 1, energyMap.getWidth())] <=
                energyMap.getPixels()[KernelFunction::convertIndex(r, minValCol1, energyMap.getWidth())]) {
                minValCol1 = c - 1;
            }
        }
        if (c + 1 < energyMap.getWidth()) {
            if (energyMap.getPixels()[KernelFunction::convertIndex(r, c + 1, energyMap.getWidth())] <
                energyMap.getPixels()[KernelFunction::convertIndex(r, minValCol1, energyMap.getWidth())]) {
                minValCol1 = c + 1;
            }
        }
        seam[r] = minValCol1;
    }

    for (int r = int(energyMap.getHeight() / 2 + 2); r < energyMap.getHeight(); ++r) {
        auto c = minValCol2;
        if (c > 0) {
            if (energyMap.getPixels()[KernelFunction::convertIndex(r, c - 1, energyMap.getWidth())] <=
                energyMap.getPixels()[KernelFunction::convertIndex(r, minValCol2, energyMap.getWidth())]) {
                minValCol2 = c - 1;
            }
        }
        if (c + 1 < energyMap.getWidth()) {
            if (energyMap.getPixels()[KernelFunction::convertIndex(r, c + 1, energyMap.getWidth())] <
                energyMap.getPixels()[KernelFunction::convertIndex(r, minValCol2, energyMap.getWidth())]) {
                minValCol2 = c + 1;
            }
        }
        seam[r] = minValCol2;
    }
}
