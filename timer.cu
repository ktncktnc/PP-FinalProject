#include "timer.cuh"

GpuTimer::GpuTimer() {
    hipEventCreate(&start);
    hipEventCreate(&stop);
}

GpuTimer::~GpuTimer() {
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void GpuTimer::Start() {
    hipEventRecord(start, 0);
    hipEventSynchronize(start);
}

void GpuTimer::Stop() {
    hipEventRecord(stop, 0);
}

float GpuTimer::Elapsed() {
    float elapsed;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
}