#include "hip/hip_runtime.h"
#include "parallel_solution_v4.cuh"
#include "timer.cuh"

namespace KernelFunction {
    __device__ u_int32_t blockCount = 0;

    __global__ void
    updateSeamMapKernelPipelining(int32_t *input, u_int32_t inputWidth,
                                  bool volatile *isBlockFinished) {
        // 1.  Get block Index
        __shared__ u_int32_t newBlockIdx;
        if (threadIdx.x == 0) {
            newBlockIdx = atomicAdd(&blockCount, 1);
        }
        __syncthreads();

        u_int32_t numBlocksPerRow = ((inputWidth - 1) / blockDim.x + 1);
        u_int32_t currentRow = (newBlockIdx / numBlocksPerRow) + 1;
        u_int32_t currentRowBlock = newBlockIdx % numBlocksPerRow;
        u_int32_t c = currentRowBlock * blockDim.x + threadIdx.x;
        int32_t minVal = 0;

        // 2. Waiting for before block newBlockIdx - numBlocksPerRow
        if (threadIdx.x == 0) {
            if (newBlockIdx >= numBlocksPerRow) {
                while (!isBlockFinished[newBlockIdx - numBlocksPerRow]);
            }
            __threadfence();
        }
        __syncthreads();

        if (c < inputWidth && threadIdx.x != blockDim.x - 1 && threadIdx.x != 0) {
            minVal = input[convertIndex(currentRow - 1, c, inputWidth)];
            if (c > 0)
                minVal = min(minVal, input[convertIndex(currentRow - 1, c - 1, inputWidth)]);
            if (c + 1 < inputWidth)
                minVal = min(minVal, input[convertIndex(currentRow - 1, c + 1, inputWidth)]);
        }

        // 3. Waiting for before block newBlockIdx + 1 and newBlockIdx - 1
        if (threadIdx.x == 0 || threadIdx.x == blockDim.x - 1) {
            if (newBlockIdx > numBlocksPerRow) {
                if (newBlockIdx % numBlocksPerRow != 0) {
                    while (!isBlockFinished[newBlockIdx - numBlocksPerRow - 1]);
                }
                if (newBlockIdx % numBlocksPerRow != numBlocksPerRow - 1) {
                    while (!isBlockFinished[newBlockIdx - numBlocksPerRow + 1]);
                }
                __threadfence();
            }
            if (c < inputWidth) {
                minVal = input[convertIndex(currentRow - 1, c, inputWidth)];
                if (c > 0)
                    minVal = min(minVal, input[convertIndex(currentRow - 1, c - 1, inputWidth)]);
                if (c + 1 < inputWidth)
                    minVal = min(minVal, input[convertIndex(currentRow - 1, c + 1, inputWidth)]);
            }
        }

        if (c < inputWidth)
            input[convertIndex(currentRow, c, inputWidth)] += minVal;

        __syncthreads();

        // 4. Mark Threads as Done
        __threadfence();
        if (threadIdx.x == 0) {
            isBlockFinished[newBlockIdx] = true;
        }
    }
}


PnmImage ParallelSolutionV4::run(const PnmImage &inputImage, int argc, char **argv) {

    // Extract arguments
    int nDeletingSeams = 1;
    dim3 blockSize(32, 32); // Default
    if (argc > 0)
        nDeletingSeams = int(strtol(argv[0], nullptr, 10));
    if (argc > 1) {
        blockSize.x = strtol(argv[1], nullptr, 10);
        blockSize.y = strtol(argv[2], nullptr, 10);
    }

    // Start Timer
    printf("Running Parallel Solution Version 4 with blockSize=(%d;%d).\n", blockSize.x, blockSize.y);
    GpuTimer timer;
    timer.Start();

    // Create Host Variable
    PnmImage outputImage(inputImage.getWidth() - nDeletingSeams, inputImage.getHeight());

    // Create Host Memory
    auto *seam = (uint32_t *) malloc(inputImage.getHeight() * sizeof(uint32_t));
    auto *energyMap = (int32_t *) malloc(inputImage.getHeight() * inputImage.getWidth() * sizeof(int32_t));

    // Create Device Memory
    uchar3 *d_inputImage;
    CHECK(hipMalloc(&d_inputImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    uchar3 *d_inputImageTemp;
    CHECK(hipMalloc(&d_inputImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    int32_t *d_grayImage;
    CHECK(hipMalloc(&d_grayImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_grayImageTemp;
    CHECK(hipMalloc(&d_grayImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_energyMap;
    CHECK(hipMalloc(&d_energyMap, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_filterX;
    CHECK(hipMalloc(&d_filterX, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))
    int32_t *d_filterY;
    CHECK(hipMalloc(&d_filterY, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpy(d_inputImage, inputImage.getPixels(),
                     inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3), hipMemcpyHostToDevice))
    CHECK(hipMemcpy(d_filterX, SOBEL_X, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t), hipMemcpyHostToDevice))
    CHECK(hipMemcpy(d_filterY, SOBEL_Y, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t), hipMemcpyHostToDevice))

    // Run Kernel functions
    convertToGrayScale(d_inputImage, inputImage.getWidth(), inputImage.getHeight(), blockSize, d_grayImage);
    for (int i = 0; i < nDeletingSeams; ++i) {
        // 1. Calculate the Energy Map
        calculateEnergyMap(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), d_filterX, d_filterY,
                           FILTER_SIZE, blockSize, d_energyMap);
        // 2. Dynamic Programming
        calculateSeamMap(d_energyMap, inputImage.getWidth() - i, inputImage.getHeight(), blockSize.x * blockSize.y);
        // 3. Extract the seam
        CHECK(hipMemcpy(energyMap, d_energyMap,
                         (inputImage.getWidth() - i) * inputImage.getHeight() * sizeof(int32_t),
                         hipMemcpyDeviceToHost))
        extractSeam(energyMap, inputImage.getWidth() - i, inputImage.getHeight(), seam);
        // 4. Delete the seam
        deleteSeam(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_grayImageTemp);
        deleteSeam(d_inputImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_inputImageTemp);
        swap(d_grayImage, d_grayImageTemp);
        swap(d_inputImage, d_inputImageTemp);
    }

    // Copy memory from device to host
    CHECK(hipMemcpy(outputImage.getPixels(), d_inputImage,
                     outputImage.getWidth() * outputImage.getHeight() * sizeof(uchar3), hipMemcpyDeviceToHost))

    // Free Device Memory
    CHECK(hipFree(d_inputImage))
    CHECK(hipFree(d_inputImageTemp))
    CHECK(hipFree(d_grayImage))
    CHECK(hipFree(d_grayImageTemp))
    CHECK(hipFree(d_energyMap))
    CHECK(hipFree(d_filterX))
    CHECK(hipFree(d_filterY))

    // Free Host Memory
    free(seam);
    free(energyMap);

    // Stop Timer
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
    printf("-------------------------------\n");

    // Return
    return outputImage;
}

void ParallelSolutionV4::calculateSeamMap(int32_t *d_inputImage, uint32_t inputWidth, uint32_t inputHeight,
                                          uint32_t blockSize) {
    uint32_t gridSize = ((inputWidth - 1) / blockSize + 1) * (inputHeight - 1);
    IntImage outputImage = IntImage(inputWidth, inputHeight);
    uint32_t zero = 0;

    // Create Device Memory
    bool *isBlockFinished;
    CHECK(hipMalloc(&isBlockFinished, gridSize * sizeof(bool)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::blockCount), &zero, sizeof(u_int32_t), 0, hipMemcpyHostToDevice))
    CHECK(hipMemset(isBlockFinished, 0, gridSize * sizeof(bool)))

    // Run Device Methods
    KernelFunction::updateSeamMapKernelPipelining<<<gridSize, blockSize>>>(d_inputImage, inputWidth, isBlockFinished);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    // Copy Memory from Device to Host

    // Free Device Memory
    CHECK(hipFree(isBlockFinished))

    // Free Host Memory

    // Return result
}
