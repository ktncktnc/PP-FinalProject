#include "hip/hip_runtime.h"
#include "parallel_solution_v4.cuh"
#include "timer.cuh"

namespace KernelFunction {
    __device__ u_int32_t blockCount;
    __global__ void
    updateSeamMapKernelPipelining(int32_t *input, u_int32_t inputWidth,
                                  bool volatile *isBlockFinished) {
        // 1.  Get block Index
        __shared__ u_int32_t newBlockIdx;
        if (threadIdx.x == 0) {
            newBlockIdx = atomicAdd(&blockCount, 1);
        }
        __syncthreads();

        u_int32_t numBlocksPerRow = ((inputWidth - 1) / blockDim.x + 1);
        u_int32_t currentRow = (newBlockIdx / numBlocksPerRow) + 1;
        u_int32_t currentRowBlock = newBlockIdx % numBlocksPerRow;
        u_int32_t c = currentRowBlock * blockDim.x + threadIdx.x;
        int32_t minVal = 0;

        // 2. Waiting for before block newBlockIdx - numBlocksPerRow
        if (threadIdx.x == 0) {
            if (newBlockIdx >= numBlocksPerRow) {
                while (!isBlockFinished[newBlockIdx - numBlocksPerRow]);
            }
            __threadfence();
        }
        __syncthreads();

        if (c < inputWidth && threadIdx.x != blockDim.x - 1 && threadIdx.x != 0) {
            minVal = input[convertIndex(currentRow - 1, c, inputWidth)];
            if (c > 0)
                minVal = min(minVal, input[convertIndex(currentRow - 1, c - 1, inputWidth)]);
            if (c + 1 < inputWidth)
                minVal = min(minVal, input[convertIndex(currentRow - 1, c + 1, inputWidth)]);
        }

        // 3. Waiting for before block newBlockIdx + 1 and newBlockIdx - 1
        if (threadIdx.x == 0 || threadIdx.x == blockDim.x - 1) {
            if (newBlockIdx > numBlocksPerRow) {
                if (newBlockIdx % numBlocksPerRow != 0) {
                    while (!isBlockFinished[newBlockIdx - numBlocksPerRow - 1]);
                }
                if (newBlockIdx % numBlocksPerRow != numBlocksPerRow - 1) {
                    while (!isBlockFinished[newBlockIdx - numBlocksPerRow + 1]);
                }
                __threadfence();
            }
            if (c < inputWidth) {
                minVal = input[convertIndex(currentRow - 1, c, inputWidth)];
                if (c > 0)
                    minVal = min(minVal, input[convertIndex(currentRow - 1, c - 1, inputWidth)]);
                if (c + 1 < inputWidth)
                    minVal = min(minVal, input[convertIndex(currentRow - 1, c + 1, inputWidth)]);
            }
        }

        if (c < inputWidth)
            input[convertIndex(currentRow, c, inputWidth)] += minVal;

        __syncthreads();

        // 4. Mark Threads as Done
        __threadfence();
        if (threadIdx.x == 0) {
            isBlockFinished[newBlockIdx] = true;
        }
    }
}


PnmImage ParallelSolutionV4::run(const PnmImage &inputImage, int argc, char **argv) {

    // Extract arguments
    int nDeletingSeams = 1;
    dim3 blockSize(32, 32); // Default
    if (argc > 0)
        nDeletingSeams = int(strtol(argv[0], nullptr, 10));
    if (argc > 1) {
        blockSize.x = strtol(argv[1], nullptr, 10);
        blockSize.y = strtol(argv[2], nullptr, 10);
    }

    // Start Timer
    printf("Running Parallel Solution Version 4 with blockSize=(%d;%d).\n", blockSize.x, blockSize.y);
    GpuTimer timer;
    GpuTimer stepTimer;

    float cal_energy_time = 0;
    float cal_seam_time = 0;
    float extract_seam_time = 0;
    float delete_seam_time = 0;

    timer.Start();

    // Create Host Variable
    PnmImage outputImage(inputImage.getWidth() - nDeletingSeams, inputImage.getHeight());

    // Create Host Memory
    auto *seam = (uint32_t *) malloc(inputImage.getHeight() * sizeof(uint32_t));
    auto *energyMap = (int32_t *) malloc(inputImage.getHeight() * inputImage.getWidth() * sizeof(int32_t));

    // Create Device Memory
    uchar3 *d_inputImage;
    CHECK(hipMalloc(&d_inputImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    uchar3 *d_inputImageTemp;
    CHECK(hipMalloc(&d_inputImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    int32_t *d_grayImage;
    CHECK(hipMalloc(&d_grayImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_grayImageTemp;
    CHECK(hipMalloc(&d_grayImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_energyMap;
    CHECK(hipMalloc(&d_energyMap, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_filterX;
    CHECK(hipMalloc(&d_filterX, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))
    int32_t *d_filterY;
    CHECK(hipMalloc(&d_filterY, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpy(d_inputImage, inputImage.getPixels(),
                     inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3), hipMemcpyHostToDevice))
    CHECK(hipMemcpy(d_filterX, SOBEL_X, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t), hipMemcpyHostToDevice))
    CHECK(hipMemcpy(d_filterY, SOBEL_Y, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t), hipMemcpyHostToDevice))

    // Run Kernel functions
    convertToGrayScale(d_inputImage, inputImage.getWidth(), inputImage.getHeight(), blockSize, d_grayImage);
    for (int i = 0; i < nDeletingSeams; ++i) {
        // 1. Calculate the Energy Map
        stepTimer.Start();
        calculateEnergyMap(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), d_filterX, d_filterY,
                           FILTER_SIZE, blockSize, d_energyMap);
        stepTimer.Stop();
        cal_energy_time += stepTimer.Elapsed();

        // 2. Dynamic Programming
        stepTimer.Start();
        calculateSeamMap(d_energyMap, inputImage.getWidth() - i, inputImage.getHeight(), blockSize.x * blockSize.y);
        stepTimer.Stop();
        cal_seam_time += stepTimer.Elapsed();

        // 3. Extract the seam
        stepTimer.Start();
        CHECK(hipMemcpy(energyMap, d_energyMap,
                         (inputImage.getWidth() - i) * inputImage.getHeight() * sizeof(int32_t),
                         hipMemcpyDeviceToHost))
        extractSeam(energyMap, inputImage.getWidth() - i, inputImage.getHeight(), seam);
        stepTimer.Stop();
        extract_seam_time += stepTimer.Elapsed();

        // 4. Delete the seam
        stepTimer.Start();
        deleteSeam(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_grayImageTemp);
        deleteSeam(d_inputImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_inputImageTemp);
        stepTimer.Stop();
        delete_seam_time += stepTimer.Elapsed();

        swap(d_grayImage, d_grayImageTemp);
        swap(d_inputImage, d_inputImageTemp);
    }

    // Copy memory from device to host
    CHECK(hipMemcpy(outputImage.getPixels(), d_inputImage,
                     outputImage.getWidth() * outputImage.getHeight() * sizeof(uchar3), hipMemcpyDeviceToHost))

    // Free Device Memory
    CHECK(hipFree(d_inputImage))
    CHECK(hipFree(d_inputImageTemp))
    CHECK(hipFree(d_grayImage))
    CHECK(hipFree(d_grayImageTemp))
    CHECK(hipFree(d_energyMap))
    CHECK(hipFree(d_filterX))
    CHECK(hipFree(d_filterY))

    // Free Host Memory
    free(seam);
    free(energyMap);

    // Stop Timer
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
    printf("Step time: %.3f ms \t %.3f ms \t %.3f ms \t %.3f ms\n", cal_energy_time, cal_seam_time, extract_seam_time, delete_seam_time);
    printf("-------------------------------\n");

    // Return
    return outputImage;
}

void ParallelSolutionV4::calculateSeamMap(int32_t *d_inputImage, uint32_t inputWidth, uint32_t inputHeight,
                                          uint32_t blockSize) {
    uint32_t gridSize = ((inputWidth - 1) / blockSize + 1) * (inputHeight - 1);
    uint32_t zero = 0;

    // Create Device Memory
    bool *isBlockFinished;
    CHECK(hipMalloc(&isBlockFinished, gridSize * sizeof(bool)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::blockCount), &zero, sizeof(u_int32_t), 0, hipMemcpyHostToDevice))
    CHECK(hipMemset(isBlockFinished, 0, gridSize * sizeof(bool)))

    // Run Device Methods
    KernelFunction::updateSeamMapKernelPipelining<<<gridSize, blockSize>>>(d_inputImage, inputWidth, isBlockFinished);
    hipDeviceSynchronize();
    CHECK(hipGetLastError())

    // Copy Memory from Device to Host

    // Free Device Memory
    CHECK(hipFree(isBlockFinished))

    // Free Host Memory

    // Return result
}
