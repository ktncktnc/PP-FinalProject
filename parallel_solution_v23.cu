#include "hip/hip_runtime.h"
#include "parallel_solution_v23.cuh"
#include "parallel_solution_v3.cuh"
#include "timer.cuh"

void ParallelSolutionV23::calculateSeamMap(int32_t *d_inputImage, uint32_t inputWidth, uint32_t inputHeight,
                                           uint32_t blockSize) {
    // Create Host Memory
    uint32_t gridSize = (inputWidth - 1) / blockSize + 1;

    // Create Device Memory

    // Run Device Methods
    hipStream_t streamForward, streamBackward;
    hipStreamCreate(&streamForward);
    hipStreamCreate(&streamBackward);

    for (int i = 1; i < inputHeight / 2; ++i) {
        // Forward
        KernelFunction::updateSeamMapKernel<<<gridSize, blockSize, 0, streamForward>>>(d_inputImage, inputWidth, i);
        // Backward
        if (int(inputHeight) - i - 1 >= inputHeight / 2) {
            KernelFunction::updateSeamMapKernelBackward<<<gridSize, blockSize, 0, streamBackward>>>(d_inputImage, inputWidth,
                    int(inputHeight) - i - 1);
        }
        hipStreamSynchronize(streamForward);
        hipStreamSynchronize(streamBackward);
        CHECK(hipGetLastError())
    }

    if (inputHeight % 2 == 1) {
        KernelFunction::updateSeamMapKernelBackward<<<gridSize, blockSize, 0, streamBackward>>>(d_inputImage, inputWidth,
                int(inputHeight) - int(inputHeight) / 2 - 1);
        hipStreamSynchronize(streamForward);
        hipStreamSynchronize(streamBackward);
        CHECK(hipGetLastError())
    }

    hipStreamDestroy(streamForward);
    hipStreamDestroy(streamBackward);

    // Copy Memory from Device to Host

    // Free Device Memory

    // Free Host Memory

    // Return result
}

void
ParallelSolutionV23::extractSeam(const int32_t *energyMap, uint32_t inputWidth, uint32_t inputHeight, uint32_t *seam) {
    // Find minSeam
    u_int32_t minValCol1 = 0;
    u_int32_t minValCol2 = 0;
    u_int32_t middleRow = inputHeight / 2 - 1;
    int32_t bestVal = energyMap[KernelFunction::convertIndex(middleRow, 0, inputWidth)] +
                      energyMap[KernelFunction::convertIndex(middleRow + 1, 0, inputWidth)];

    for (int c = 0; c < inputWidth; ++c) {
        if (energyMap[KernelFunction::convertIndex(middleRow, c, inputWidth)] +
            energyMap[KernelFunction::convertIndex(middleRow + 1, c, inputWidth)]
            < bestVal) {
            bestVal = energyMap[KernelFunction::convertIndex(middleRow, c, inputWidth)] +
                      energyMap[KernelFunction::convertIndex(middleRow + 1, c, inputWidth)];
            minValCol1 = c;
            minValCol2 = c;
        }

        if (c > 0 &&
            energyMap[KernelFunction::convertIndex(middleRow, c - 1, inputWidth)] +
            energyMap[KernelFunction::convertIndex(middleRow + 1, c, inputWidth)]
            <= bestVal) {
            bestVal = energyMap[KernelFunction::convertIndex(middleRow, c - 1, inputWidth)] +
                      energyMap[KernelFunction::convertIndex(middleRow + 1, c, inputWidth)];
            minValCol1 = c - 1;
            minValCol2 = c;
        }

        if (c + 1 < inputWidth &&
            energyMap[KernelFunction::convertIndex(middleRow, c + 1, inputWidth)] +
            energyMap[KernelFunction::convertIndex(middleRow + 1, c, inputWidth)]
            < bestVal) {
            bestVal = energyMap[KernelFunction::convertIndex(middleRow, c + 1, inputWidth)] +
                      energyMap[KernelFunction::convertIndex(middleRow + 1, c, inputWidth)];
            minValCol1 = c + 1;
            minValCol2 = c;
        }
    }
    // Trace back
    seam[inputHeight / 2 - 1] = minValCol1;
    seam[inputHeight / 2] = minValCol2;

    for (int r = int(inputHeight / 2) - 2; r >= 0; --r) {
        auto c = minValCol1;
        if (c > 0) {
            if (energyMap[KernelFunction::convertIndex(r, c - 1, inputWidth)] <=
                energyMap[KernelFunction::convertIndex(r, minValCol1, inputWidth)]) {
                minValCol1 = c - 1;
            }
        }
        if (c + 1 < inputWidth) {
            if (energyMap[KernelFunction::convertIndex(r, c + 1, inputWidth)] <
                energyMap[KernelFunction::convertIndex(r, minValCol1, inputWidth)]) {
                minValCol1 = c + 1;
            }
        }
        seam[r] = minValCol1;
    }

    for (int r = int(inputHeight / 2) + 1; r < inputHeight; ++r) {
        auto c = minValCol2;
        if (c > 0) {
            if (energyMap[KernelFunction::convertIndex(r, c - 1, inputWidth)] <=
                energyMap[KernelFunction::convertIndex(r, minValCol2, inputWidth)]) {
                minValCol2 = c - 1;
            }
        }
        if (c + 1 < inputWidth) {
            if (energyMap[KernelFunction::convertIndex(r, c + 1, inputWidth)] <
                energyMap[KernelFunction::convertIndex(r, minValCol2, inputWidth)]) {
                minValCol2 = c + 1;
            }
        }
        seam[r] = minValCol2;
    }
}

PnmImage ParallelSolutionV23::run(const PnmImage &inputImage, int argc, char **argv) {
    // Extract arguments
    int nDeletingSeams = 1;
    dim3 blockSize(32, 32); // Default
    if (argc > 0)
        nDeletingSeams = int(strtol(argv[0], nullptr, 10));
    if (argc > 1) {
        blockSize.x = strtol(argv[1], nullptr, 10);
        blockSize.y = strtol(argv[2], nullptr, 10);
    }

    // Start Timer
    printf("Running Parallel Solution Version 2 + 3 with blockSize=(%d;%d).\n", blockSize.x, blockSize.y);
    GpuTimer timer;
    GpuTimer stepTimer;

    float cal_energy_time = 0;
    float cal_seam_time = 0;
    float extract_seam_time = 0;
    float delete_seam_time = 0;

    timer.Start();

    // Create Host Variable
    PnmImage outputImage(inputImage.getWidth() - nDeletingSeams, inputImage.getHeight());

    // Create Host Memory
    auto *seam = (uint32_t *) malloc(inputImage.getHeight() * sizeof(uint32_t));
    auto *energyMap = (int32_t *) malloc(inputImage.getHeight() * inputImage.getWidth() * sizeof(int32_t));

    // Create Device Memory
    uchar3 *d_inputImage;
    CHECK(hipMalloc(&d_inputImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    uchar3 *d_inputImageTemp;
    CHECK(hipMalloc(&d_inputImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3)))
    int32_t *d_grayImage;
    CHECK(hipMalloc(&d_grayImage, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_grayImageTemp;
    CHECK(hipMalloc(&d_grayImageTemp, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))
    int32_t *d_energyMap;
    CHECK(hipMalloc(&d_energyMap, inputImage.getWidth() * inputImage.getHeight() * sizeof(int32_t)))

    // Copy Memory from Host to Device
    CHECK(hipMemcpy(d_inputImage, inputImage.getPixels(),
                     inputImage.getWidth() * inputImage.getHeight() * sizeof(uchar3), hipMemcpyHostToDevice))

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::c_filterX), SOBEL_X, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(KernelFunction::c_filterY), SOBEL_Y, FILTER_SIZE * FILTER_SIZE * sizeof(int32_t)))

    // Run Kernel functions
    convertToGrayScale(d_inputImage, inputImage.getWidth(), inputImage.getHeight(), blockSize, d_grayImage);
    for (int i = 0; i < nDeletingSeams; ++i) {
        // 1. Calculate the Energy Map
        stepTimer.Start();
        calculateEnergyMap(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), blockSize, d_energyMap);
        stepTimer.Stop();
        cal_energy_time += stepTimer.Elapsed();

        // 2. Dynamic Programming
        stepTimer.Start();
        calculateSeamMap(d_energyMap, inputImage.getWidth() - i, inputImage.getHeight(), blockSize.x * blockSize.y);
        stepTimer.Stop();
        cal_seam_time += stepTimer.Elapsed();

        // 3. Extract the seam
        stepTimer.Start();
        CHECK(hipMemcpy(energyMap, d_energyMap,
                         (inputImage.getWidth() - i) * inputImage.getHeight() * sizeof(int32_t),
                         hipMemcpyDeviceToHost))
        extractSeam(energyMap, inputImage.getWidth() - i, inputImage.getHeight(), seam);
        stepTimer.Stop();
        extract_seam_time += stepTimer.Elapsed();

        // 4. Delete the seam
        stepTimer.Start();
        deleteSeam(d_grayImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_grayImageTemp);
        deleteSeam(d_inputImage, inputImage.getWidth() - i, inputImage.getHeight(), seam, blockSize, d_inputImageTemp);
        stepTimer.Stop();
        delete_seam_time += stepTimer.Elapsed();

        swap(d_grayImage, d_grayImageTemp);
        swap(d_inputImage, d_inputImageTemp);
    }

    // Copy memory from device to host
    CHECK(hipMemcpy(outputImage.getPixels(), d_inputImage,
                     outputImage.getWidth() * outputImage.getHeight() * sizeof(uchar3), hipMemcpyDeviceToHost))

    // Free Device Memory
    CHECK(hipFree(d_inputImage))
    CHECK(hipFree(d_inputImageTemp))
    CHECK(hipFree(d_grayImage))
    CHECK(hipFree(d_grayImageTemp))
    CHECK(hipFree(d_energyMap))

    // Free Host Memory
    free(seam);
    free(energyMap);

    // Stop Timer
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
    printf("Step time: 2) %.3f ms \t 3) %.3f ms \t 4) %.3f ms \t 5) %.3f ms\n", cal_energy_time, cal_seam_time, extract_seam_time, delete_seam_time);
    printf("-------------------------------\n");

    // Return
    return outputImage;
}
