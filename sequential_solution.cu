#include "hip/hip_runtime.h"
#include "sequential_solution.cuh"
#include <iostream>

using namespace std;

namespace SequentialFunction {
    void scan(const int *input, u_int32_t inputWidth, u_int32_t inputHeight, const int3 *filter,
              u_int32_t filterSize, int* output) {
        int index, k_index, k_x, k_y, k_value, sum;

        //For each pixel in image
        for(int x = 0; x < inputHeight; x++){
            for(int y = 0; y < inputWidth; y++){
                sum = 0;
                index = x*inputWidth + y;

                //For each value in kernel
                for(int i = -(filterSize/2); i <= filterSize/2; i++){
                    for(int j = -(filterSize/2); j <= filterSize/2; j++){
                        k_x = min(max(x + i, 0), inputHeight - 1);
                        k_y = min(max(y + j, 0), inputWidth - 1);

                        k_index = k_x*inputWidth + k_y;

                        k_value = (j % 3 == 0)*filter[i].x + (j % 3 == 1)*filter[i].y + (j % 3 == 2)* filter[i].z;
                        sum += input[index] * value;
                    }
                }
                output[index] = sum;
            }
        }
    }

    void convertToGray(PnmImage &inputImage, int* output){
        uchar3 *input = inputImage.getPixels();

        for(int i = 0; i < inputImage.getWidth() * inputImage.getHeight(); i++){
            output[i] = int(0.299f*input[i].x) + int(0.587f*input[i].y) + int(0.114f*input[i].z);
        }
    }

    void addAbs(const int *input_1, const int* input_2, u_int32_t inputWidth, u_int32_t inputHeight,
                 int *output) {
        int index;

        for(int x = 0; x < inputHeight; x++){
            for(int y = 0; y < inputWidth; y++) {
                index = x*inputWidth + y;
                int value = sqrt(pow(input_1[index], 2) + pow(input_2[index], 2)));
                output[index] = value;
            }
        }
    }

    void createCumulativeEnergyMap(
            const int* input, //Gradient image
            u_int32_t inputWidth,
            u_int32_t inputHeight,
            bool direction //Direction: 0: vertical 1: horizontal,
            long* output
            ){

        int a, b, c;

        // Copy first line
        if (direction == 0){
            memcpy(output, input, inputWidth * sizeof(int));
        }
        else{
            for(int i = 0; i < inputHeight; i++){
                output[i* inputWidth] = input[i * inputWidth];
            }
        }

        if (direction == 0){
            for(int row = 1; row < inputHeight; row++){
                for(int col = 0; col < inputWidth; col++){
                    a = output[(row - 1)*inputWidth + max(col - 1, 0)];
                    b = output[(row - 1)*inputWidth + col];
                    c = output[(row - 1)*inputWidth + min(col + 1, inputWidth - 1)];

                    output[row*inputWidth + col] = min(min(a, b), c);
                }
            }
        }
        else
            for(int col = 1; col < inputWidth; col++){
                for(int row = 0; row < inputHeight; row++){
                    a = output[max(row - 1, 0)*inputWidth + col - 1];
                    b = output[row*inputWidth + col - 1];
                    c = output[min(row + 1, inputHeight - 1)*inputWidth + col - 1];

                    output[row*inputWidth + col] = min(min(a, b), c);
                }
            }
        }

    void findSeamCurve(
            const long* input,
            bool direction,
            u_int32_t inputWidth,
            u_int32_t inputHeight,
            int* output
            ){
        if (direction == 0){

        }
    }
}

const int32_t ParallelSolutionBaseline::SOBEL_X[3][3] = {{1, 0, -1},
                                                         {2, 0, -2},
                                                         {1, 0, -1}};
const int32_t ParallelSolutionBaseline::SOBEL_Y[3][3] = {{1,  2,  1},
                                                         {0,  0,  0},
                                                         {-1, -2, -1}};

PnmImage SequentialSolution::run(const PnmImage &inputImage, int argc, char **argv) {
    return BaseSolution::run(inputImage, argc, argv);
}

IntImage SequentialSolution::scan(const PnmImage &inputImage) {
    int* grayImg = (int*)malloc(inputImage.getHeight() * inputImage.getWidth());

    int* grImgX, *grImgY, *grImg;
    grImgX = (int*)malloc(inputImage.getHeight() * inputImage.getWidth());
    grImgY = (int*)malloc(inputImage.getHeight() * inputImage.getWidth());
    grImg = (int*)malloc(inputImage.getHeight() * inputImage.getWidth());;

    //RGB to gray
    SequentialFunction::toGray(inputImage, grayImg);

    //Scan
    SequentialFunction::scan(grayImg, inputImage.getWidth(), inputImage.getHeight(), SOBEL_X, 3,grImgX);
    SequentialFunction::scan(grayImg, inputImage.getWidth(), inputImage.getHeight(), SOBEL_X, 3,grImgY);

    SequentialFunction::addAbs(imgX, imgY, inputImage.getWidth(), inputImage.getHeight(), grImg);

    IntImage outputImage = IntImage(inputImage.getWidth(), inputImage.getHeight());


}